#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <stdlib.h>

int bruteIncrement(char* brute, int alphabetLen, int wordLen, int incrementBy) {
	int i = 0;
	while(incrementBy > 0 && i < wordLen) {
		int add = incrementBy + brute[i];
		brute[i] = (char)(add % alphabetLen);
		incrementBy = add / alphabetLen;
		i++;
	}
	return incrementBy == 0; 
}


void bruteToString(char *brute, int wordLen, char *alphabet, char *out){
	
	for(int i=0;i<wordLen;i++) {
		out[i]=alphabet[brute[i]];
	}
	out[wordLen]='\0';
}


int any(char *list, int listSize){
	for(int i=0;i<listSize;i++){
		if(list[i])return 1;
	}
	return 0;
}


void printWork(char *work, int workLen){
	char *out[] = { "false", "true" };
	for(int i=0;i<workLen;i++){
		printf("%s ", out[work[i]]);
	}
	printf("\n");
}

int searchPart(char *targetString, char *alphabet, char *brute, int workSize, int workerId, int wordLen, int alphabetLen){
	// Go to the start of my work
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	if (!bruteIncrement(brute, alphabetLen, wordLen, workSize*workerId)) {
        	return 0;
	}
	int count = 0;
	char out[wordLen + 1];
	while(1) {
		if(count>=workSize)break;
		bruteToString(brute, wordLen, alphabet, out);
		if(strcmp(out, targetString)==0)return 1;
		count +=1;
		if(!bruteIncrement(brute, alphabetLen, wordLen, 1))break;
	}
	return 0;
}

__global__ int search() {
	
}
// not in cuda
int search(char *targetString, char *alphabet, int numWorkers, int workSize){
	int wordLen = strlen(targetString);
	int alphabetLen = strlen(alphabet);

	char brute[wordLen];	
	
	for(int i = 0; i < wordLen; i++) {
		brute[i] = 0;
	}

	while(1){
		char work[numWorkers];
		for(int workerId=0; workerId < numWorkers; workerId++) {
			char brutePart[wordLen]; // Really probably need another brutePart for each worker
			for(int i=0; i < wordLen; i++) {
				brutePart[i]=brute[i];
			}
			work[workerId]=searchPart(targetString, alphabet, brutePart, workSize, workerId, wordLen, alphabetLen);
		}
		// printWork(work, numWorkers);
		if(any(work, numWorkers))return 1;
	
		// advance to the next major chunk of work
		if(!bruteIncrement(brute, alphabetLen, wordLen, workSize*numWorkers)){
			break;
		}
	}
	return 0;
}

int main( int argc, char** argv) {
	char *targetString = argv[1];
	char *alphabet = argv[2];
	int numWorkers = atoi(argv[3]);
	int workSize = atoi(argv[4]);

	if(search(targetString, alphabet, numWorkers, workSize)){
		printf("found\n");
	} else {
		printf("notfound\n");
	}
}
