
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string.h>
#include <stdlib.h>

__device__ __host__ int bruteIncrement(char* brute, int alphabetLen, int wordLen, int incrementBy) {
	int i = 0;
	while(incrementBy > 0 && i < wordLen) {
		int add = incrementBy + brute[i];
		brute[i] = (char)(add % alphabetLen);
		incrementBy = add / alphabetLen;
		i++;
	}
	return incrementBy == 0; 
}

__device__ void bruteToString(char *brute, int wordLen, char *alphabet, char *out){
	for(int i=0;i<wordLen;i++){
		out[i]=alphabet[brute[i]];
	}
	out[wordLen]='\0';
}

__device__ int any(char *list, int listSize){
	for(int i=0;i<listSize;i++){
		if(list[i])return 1;
	}
	return 0;
}

void printWork(char *work, int workLen){
	char *out[] = { "false", "true" };
	for(int i=0;i<workLen;i++){
		printf("%s ", out[work[i]]);
	}
	printf("\n");
}


__device__ int my_strcmp(char a[], char b[])
{
   int c = 0;

   while (a[c] == b[c]) {
      if (a[c] == '\0' || b[c] == '\0')
         break;
      c++;
   }

   if (a[c] == '\0' && b[c] == '\0')
      return 0;
   else
      return -1;
}
__device__ int searchPart(char *targetString, char *alphabet, char *brute, int workSize, int workerId, int wordLen, int alphabetLen){
	// Go to the start of my work
	if(!bruteIncrement(brute, alphabetLen, wordLen, workSize*workerId)){
        	return 0;
        }
        int count = 0;
	char *out = new char[wordLen + 1];
        while(1){
		if(count>=workSize)break;
		bruteToString(brute, wordLen, alphabet, out);
		if(my_strcmp(out, targetString)==0)return 1;
		count +=1;
		if(!bruteIncrement(brute, alphabetLen, wordLen, 1))break;
        }
        return 0;
}

__global__ void search(int *foundWord, char *targetString, char *alphabet, int wordLen, int alphabetLen){
	long wordId = threadIdx.x + (blockIdx.x * blockDim.x);
	char *brute = new char[wordLen]; 
	for(int i=0;i<wordLen;i++)brute[i]=0;
	
	char *brutePart = new char[wordLen]; // Really probably need another brutePart for each worker
	for(int i=0;i<wordLen;i++)brutePart[i]=brute[i];
	int success = searchPart(targetString, alphabet, brutePart, 1024, wordId, wordLen, alphabetLen);
	if (success == 1){
		*foundWord = 1;
	}
	return;
}


int main( int argc, char** argv) {
	char *targetString = argv[1];
	char *alphabet = argv[2];

	int wordLen = strlen(targetString);
	int alphabetLen = strlen(alphabet);


	
	
	char *targetString_device;
	char *alphabet_device;
	int *foundWord_device = 0;

	hipMallocManaged(&foundWord_device, sizeof(int));
	hipMalloc((void **)&targetString_device, wordLen*sizeof(char));
	hipMalloc((void **)&alphabet_device, alphabetLen*sizeof(char));


	hipMemcpy(targetString_device, targetString, wordLen*sizeof(char), hipMemcpyHostToDevice);
	hipMemcpy(alphabet_device, alphabet, alphabetLen*sizeof(char), hipMemcpyHostToDevice);

	search<<<1024, 1024>>>(foundWord_device, targetString_device, alphabet_device, wordLen, alphabetLen);
	hipDeviceSynchronize();
	
	if(*foundWord_device == 1){
		printf("found\n");
	} else {
		printf("not found\n");
	}
	hipFree(foundWord_device);
	hipFree(targetString_device);
	hipFree(alphabet_device);
	
}

